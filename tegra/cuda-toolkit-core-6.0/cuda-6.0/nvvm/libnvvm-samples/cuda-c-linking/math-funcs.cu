
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple implementation of Mandelbrot set from Wikipedia
// http://en.wikipedia.org/wiki/Mandelbrot_set

// Note that this kernel is meant to be a simple, straight-forward
// implementation, and so may not represent optimized GPU code.
extern "C"
__device__
void mandelbrot(float* Data) {

  // Which pixel am I?
  unsigned DataX = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned DataY = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned Width = gridDim.x * blockDim.x;
  unsigned Height = gridDim.y * blockDim.y;

  float R, G, B, A;

  // Scale coordinates to (-2.5, 1) and (-1, 1)

  float NormX = (float)DataX / (float)Width;
  NormX *= 3.5f;
  NormX -= 2.5f;

  float NormY = (float)DataY / (float)Height;
  NormY *= 2.0f;
  NormY -= 1.0f;

  float X0 = NormX;
  float Y0 = NormY;

  float X = 0.0f;
  float Y = 0.0f;

  unsigned Iter = 0;
  unsigned MaxIter = 1000;

  // Iterate
  while(X*X + Y*Y < 4.0f && Iter < MaxIter) {
    float XTemp = X*X - Y*Y + X0;
    Y = 2.0f*X*Y + Y0;

    X = XTemp;

    Iter++;
  }

  unsigned ColorG = Iter % 50;
  unsigned ColorB = Iter % 25;

  R = 0.0f;
  G = (float)ColorG / 50.0f;
  B = (float)ColorB / 25.0f;
  A = 1.0f;

  Data[DataY*Width*4+DataX*4+0] = R;
  Data[DataY*Width*4+DataX*4+1] = G;
  Data[DataY*Width*4+DataX*4+2] = B;
  Data[DataY*Width*4+DataX*4+3] = A;
}
